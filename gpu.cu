#include <stdio.h>
#include <hip/hip_runtime.h>
#include <unistd.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdarg.h>
#include <png.h>


void abort_(const char * s, ...)
{
        va_list args;
        va_start(args, s);
        vfprintf(stderr, s, args);
        fprintf(stderr, "\n");
        va_end(args);
        abort();
}


int x, y;
int width, height;
png_byte color_type;
png_byte bit_depth;
png_structp png;
png_infop info;
int number_of_passes;
png_bytep *rowPointer;
png_bytep *rowPointer2;
png_bytep *rowPointer3;



void read_png_file(char* file_name)
{
        char header[8];

        FILE *fp = fopen(file_name, "rb");
        if (!fp)
                abort_("[read_png_file] File %s could not be opened for reading", file_name);
        fread(header, 1, 8, fp);


        png = png_create_read_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png)
                abort_("[read_png_file] png_create_read_struct failed");

        info = png_create_info_struct(png);
        if (!info)
                abort_("[read_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png)))
                abort_("[read_png_file] Error during init_io");


        png_init_io(png, fp);
        png_set_sig_bytes(png, 8);
        png_read_info(png, info);
        width = png_get_image_width(png, info);
        height = png_get_image_height(png, info);
        color_type = png_get_color_type(png, info);
        bit_depth = png_get_bit_depth(png, info);
        number_of_passes = png_set_interlace_handling(png);
        png_read_update_info(png, info);

        if (setjmp(png_jmpbuf(png)))
                abort_("[read_png_file] Error during read_image");

        rowPointer = (png_bytep*) malloc(sizeof(png_bytep) * height);
        for (y=0; y<height; y++)
                rowPointer[y] = (png_byte*) malloc(png_get_rowbytes(png,info));

        png_read_image(png, rowPointer);
        fclose(fp);}

void write_png_file(char* file_name)
{
        FILE *fp = fopen(file_name, "wb");
        if (!fp)
                abort_("[write_png_file] File %s could not be opened for writing", file_name);

        png = png_create_write_struct(PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);

        if (!png)
                abort_("[write_png_file] png_create_write_struct failed");

        info = png_create_info_struct(png);
        if (!info)
                abort_("[write_png_file] png_create_info_struct failed");

        if (setjmp(png_jmpbuf(png)))
                abort_("[write_png_file] Error during init_io");

        png_init_io(png, fp);

        if (setjmp(png_jmpbuf(png)))
                abort_("[write_png_file] Error during writing header");

        png_set_IHDR(png, info, width, height,
                     bit_depth, color_type, PNG_INTERLACE_NONE,
                     PNG_COMPRESSION_TYPE_BASE, PNG_FILTER_TYPE_BASE);

        png_write_info(png, info);

        if (setjmp(png_jmpbuf(png)))
                abort_("[write_png_file] Error during writing bytes");

        png_write_image(png, rowPointer);

        if (setjmp(png_jmpbuf(png)))
                abort_("[write_png_file] Error during end of write");

        png_write_end(png, NULL);

        for (y=0; y<height; y++)
                free(rowPointer[y]);

        free(rowPointer);
        fclose(fp);}


__global__ void myBlur3( int *r_I, int *g_I,  int *b_I,
	 int totalPixels, int height, int nthrds,int begin2 , int end2, int p , int nrows , int width){
   // int i = blockDim.x * blockIdx.x + threadIdx.x;
   int idx =  threadIdx.x;
    int istart,iend;
    

	
      if ( idx!= 0 ) {
        istart = idx * height*width / nthrds;
        iend = (idx +1) * height *width/ nthrds;
        if (iend + p >= height*width){
          iend = (height*width)-p ;
        }
      }else {
        istart =begin2;
        iend = end2;
      }
	int i,aux;

    // printf("start  %i end %i width  %i height %i \n", istart, iend,width, height );

    // printf("empieza  red %d  green %d blue %d\n", r_I[368676], g_I[368676],b_I[368676] );
    // printf(" red %d  green %d blue %d\n", r_I[iend], g_I[iend],b_I[iend] );
	for( i = istart; i <  iend; i++){
		if ( i < width ){
			aux=0;	
		}
		if ( i+width >= (height*width) ){
			aux=0;	
		}	
		if(  nrows == 3){
 //printf("START  red %d  green %d blue %d\n", r_I[i], g_I[i],b_I[i] );
  
		 r_I[i] = (double)(r_I[i]+r_I[i-1]+r_I[i+1]+r_I[i-aux]+r_I[i-1-aux]+r_I[i+1-aux]+r_I[i+aux]+r_I[i-1+aux]+r_I[i+1+aux])/9;
		g_I[i] = (double)(g_I[i]+g_I[i-1]+g_I[i+1]+g_I[i-aux]+g_I[i-1-aux]+g_I[i+1-aux]+g_I[i+aux]+g_I[i-1+aux]+g_I[i+1+aux])/9;
		b_I[i] = (double)(b_I[i]+b_I[i-1]+b_I[i+1]+b_I[i-aux]+b_I[i-1-aux]+b_I[i+1-aux]+b_I[i+aux]+b_I[i-1+aux]+b_I[i+1+aux])/9;
	}
	if(  nrows == 5){
	r_I[i] = (r_I[i]+r_I[i-1]+r_I[i-2]+r_I[i+1]+r_I[i+2]+
            r_I[i-aux]+r_I[i-1-aux]+r_I[i-2-aux]+r_I[i+1-aux]+r_I[i+2-aux]+
            r_I[i-(aux*2)]+r_I[i-1-(aux*2)]+r_I[i-2-(aux*2)]+r_I[i+1-(aux*2)]+r_I[i+2-(aux*2)]+
            r_I[i+aux]+r_I[i-1+aux]+r_I[i-2+aux]+r_I[i+1+aux]+r_I[i+2+aux]+
            r_I[i+(aux*2)]+r_I[i-1+(aux*2)]+r_I[i-2+(aux*2)]+r_I[i+1+(aux*2)]+r_I[i+2+(aux*2)])/25;
        g_I[i] = (g_I[i]+g_I[i-1]+g_I[i-2]+g_I[i+1]+g_I[i+2]+
            g_I[i-aux]+g_I[i-1-aux]+g_I[i-2-aux]+g_I[i+1-aux]+g_I[i+2-aux]+
            g_I[i-(aux*2)]+g_I[i-1-(aux*2)]+g_I[i-2-(aux*2)]+g_I[i+1-(aux*2)]+g_I[i+2-(aux*2)]+
            g_I[i+aux]+g_I[i-1+aux]+g_I[i-2+aux]+g_I[i+1+aux]+g_I[i+2+aux]+
            g_I[i+(aux*2)]+g_I[i-1+(aux*2)]+g_I[i-2+(aux*2)]+g_I[i+1+(aux*2)]+g_I[i+2+(aux*2)])/25;
        b_I[i] = (b_I[i]+b_I[i-1]+b_I[i-2]+b_I[i+1]+b_I[i+2]+
            b_I[i-aux]+b_I[i-1-aux]+b_I[i-2-aux]+b_I[i+1-aux]+b_I[i+2-aux]+
            b_I[i-(aux*2)]+b_I[i-1-(aux*2)]+b_I[i-2-(aux*2)]+b_I[i+1-(aux*2)]+b_I[i+2-(aux*2)]+
            b_I[i+aux]+b_I[i-1+aux]+b_I[i-2+aux]+b_I[i+1+aux]+b_I[i+2+aux]+
            b_I[i+(aux*2)]+b_I[i-1+(aux*2)]+b_I[i-2+(aux*2)]+b_I[i+1+(aux*2)]+b_I[i+2+(aux*2)])/25;
   	 }

	if(  nrows == 7){
	r_I[i] = (r_I[i]+r_I[i-1]+r_I[i-2]+r_I[i-3]+r_I[i+1]+r_I[i+2]+r_I[i+3]+
            r_I[i-aux]+r_I[i-1-aux]+r_I[i-2-aux]+r_I[i-3-aux]+r_I[i+1-aux]+r_I[i+2-aux]+r_I[i+3-aux]+
            r_I[i-(aux*2)]+r_I[i-1-(aux*2)]+r_I[i-2-(aux*2)]+r_I[i-3-(aux*2)]+r_I[i+1-(aux*2)]+r_I[i+2-(aux*2)]+r_I[i+3-(aux*2)]+
            r_I[i-(aux*3)]+r_I[i-1-(aux*3)]+r_I[i-2-(aux*3)]+r_I[i-3-(aux*3)]+r_I[i+1-(aux*3)]+r_I[i+2-(aux*3)]+r_I[i+3-(aux*3)]+
            r_I[i+aux]+r_I[i-1+aux]+r_I[i-2+aux]+r_I[i-3+aux]+r_I[i+1+aux]+r_I[i+2+aux]+r_I[i+3+aux]+
            r_I[i+(aux*2)]+r_I[i-1+(aux*2)]+r_I[i-2+(aux*2)]+r_I[i-3+(aux*2)]+r_I[i+1+(aux*2)]+r_I[i+2+(aux*2)]+r_I[i+3+(aux*2)]+
            r_I[i+(aux*3)]+r_I[i-1+(aux*3)]+r_I[i-2+(aux*3)]+r_I[i-3+(aux*3)]+r_I[i+1+(aux*3)]+r_I[i+2+(aux*3)]+r_I[i+3+(aux*3)])/49;
        g_I[i] = (g_I[i]+g_I[i-1]+g_I[i-2]+g_I[i-3]+g_I[i+1]+g_I[i+2]+g_I[i+3]+
            g_I[i-aux]+g_I[i-1-aux]+g_I[i-2-aux]+g_I[i-3-aux]+g_I[i+1-aux]+g_I[i+2-aux]+g_I[i+3-aux]+
            g_I[i-(aux*2)]+g_I[i-1-(aux*2)]+g_I[i-2-(aux*2)]+g_I[i-3-(aux*2)]+g_I[i+1-(aux*2)]+g_I[i+2-(aux*2)]+g_I[i+3-(aux*2)]+
            g_I[i-(aux*3)]+g_I[i-1-(aux*3)]+g_I[i-2-(aux*3)]+g_I[i-3-(aux*3)]+g_I[i+1-(aux*3)]+g_I[i+2-(aux*3)]+g_I[i+3-(aux*3)]+
            g_I[i+aux]+g_I[i-1+aux]+g_I[i-2+aux]+g_I[i-3+aux]+g_I[i+1+aux]+g_I[i+2+aux]+g_I[i+3+aux]+
            g_I[i+(aux*2)]+g_I[i-1+(aux*2)]+g_I[i-2+(aux*2)]+g_I[i-3+(aux*2)]+g_I[i+1+(aux*2)]+g_I[i+2+(aux*2)]+g_I[i+3+(aux*2)]+
            g_I[i+(aux*3)]+g_I[i-1+(aux*3)]+g_I[i-2+(aux*3)]+g_I[i-3+(aux*3)]+g_I[i+1+(aux*3)]+g_I[i+2+(aux*3)]+g_I[i+3+(aux*3)])/49;
        b_I[i] = (b_I[i]+b_I[i-1]+b_I[i-2]+b_I[i-3]+b_I[i+1]+b_I[i+2]+b_I[i+3]+
            b_I[i-aux]+b_I[i-1-aux]+b_I[i-2-aux]+b_I[i-3-aux]+b_I[i+1-aux]+b_I[i+2-aux]+b_I[i+3-aux]+
            b_I[i-(aux*2)]+b_I[i-1-(aux*2)]+b_I[i-2-(aux*2)]+b_I[i-3-(aux*2)]+b_I[i+1-(aux*2)]+b_I[i+2-(aux*2)]+b_I[i+3-(aux*2)]+
            b_I[i-(aux*3)]+b_I[i-1-(aux*3)]+b_I[i-2-(aux*3)]+b_I[i-3-(aux*3)]+b_I[i+1-(aux*3)]+b_I[i+2-(aux*3)]+b_I[i+3-(aux*3)]+
            b_I[i+aux]+b_I[i-1+aux]+b_I[i-2+aux]+b_I[i-3+aux]+b_I[i+1+aux]+b_I[i+2+aux]+b_I[i+3+aux]+
            b_I[i+(aux*2)]+b_I[i-1+(aux*2)]+b_I[i-2+(aux*2)]+b_I[i-3+(aux*2)]+b_I[i+1+(aux*2)]+b_I[i+2+(aux*2)]+b_I[i+3+(aux*2)]+
            b_I[i+(aux*3)]+b_I[i-1+(aux*3)]+b_I[i-2+(aux*3)]+b_I[i-3+(aux*3)]+b_I[i+1+(aux*3)]+b_I[i+2+(aux*3)]+b_I[i+3+(aux*3)])/49;

   	 }

	if(  nrows == 9){
			


	r_I[i] = ( r_I[i]+r_I[i-1]+r_I[i-2]+r_I[i-3]+r_I[i-4]+r_I[i+1]+r_I[i+2]+r_I[i+3]+r_I[i+4]+
            r_I[i-aux]+r_I[i-1-aux]+r_I[i-2-aux]+r_I[i-3-aux]+r_I[i-4-aux]+r_I[i+1-aux]+r_I[i+2-aux]+r_I[i+3-aux]+ r_I[i+4-aux]+
     r_I[i-(aux*2)]+r_I[i-1-(aux*2)]+r_I[i-2-(aux*2)]+r_I[i-3-(aux*2)]+r_I[i-4-(aux*2)]+r_I[i+1-(aux*2)]+r_I[i+2-(aux*2)]+r_I[i+3-(aux*2)]+r_I[i+4-(aux*2)]+
            r_I[i-(aux*3)]+r_I[i-1-(aux*3)]+r_I[i-2-(aux*3)]+r_I[i-3-(aux*3)]+r_I[i-4-(aux*3)]+r_I[i+1-(aux*3)]+r_I[i+2-(aux*3)]+r_I[i+3-(aux*3)]+r_I[i+4-(aux*3)]+
            r_I[i+aux]+r_I[i-1+aux]+r_I[i-2+aux]+r_I[i-3+aux]+r_I[i-4+aux]+r_I[i+1+aux]+r_I[i+2+aux]+r_I[i+3+aux]+r_I[i+4+aux]+
            r_I[i+(aux*2)]+r_I[i-1+(aux*2)]+r_I[i-2+(aux*2)]+r_I[i-3+(aux*2)]+r_I[i-4+(aux*2)]+r_I[i+1+(aux*2)]+r_I[i+2+(aux*2)]+r_I[i+3+(aux*2)]+r_I[i+4+(aux*2)]+
            r_I[i+(aux*3)]+r_I[i-1+(aux*3)]+r_I[i-2+(aux*3)]+r_I[i-3+(aux*3)]+r_I[i-4+(aux*3)]+r_I[i+1+(aux*3)]+r_I[i+2+(aux*3)]+r_I[i+3+(aux*3)]+r_I[i+4+(aux*3)]+
 r_I[i+(aux*4)]+r_I[i-1+(aux*4)]+r_I[i-2+(aux*4)]+r_I[i-3+(aux*4)]+r_I[i-4+(aux*4)]+r_I[i+1+(aux*4)]+r_I[i+2+(aux*4)]+r_I[i+3+(aux*4)]+r_I[i+4+(aux*4)]+ r_I[i-(aux*4)]+r_I[i-1-(aux*4)]+r_I[i-2-(aux*4)]+r_I[i-3-(aux*4)]+r_I[i-4-(aux*4)]+r_I[i+1-(aux*4)]+r_I[i+2-(aux*4)]+r_I[i+3-(aux*4)]+r_I[i+4-(aux*4)]   )/81;

	g_I[i] = (g_I[i]+g_I[i-1]+g_I[i-2]+g_I[i-3]+g_I[i-4]+g_I[i+1]+g_I[i+2]+g_I[i+3]+g_I[i+4]+
            g_I[i-aux]+g_I[i-1-aux]+g_I[i-2-aux]+g_I[i-3-aux]+g_I[i-4-aux]+g_I[i+1-aux]+g_I[i+2-aux]+g_I[i+3-aux]+g_I[i+4-aux]+
            g_I[i-(aux*2)]+g_I[i-1-(aux*2)]+g_I[i-2-(aux*2)]+g_I[i-3-(aux*2)]+g_I[i-4-(aux*2)]+g_I[i+1-(aux*2)]+g_I[i+2-(aux*2)]+g_I[i+3-(aux*2)]+g_I[i+4-(aux*2)]+
            g_I[i-(aux*3)]+g_I[i-1-(aux*3)]+g_I[i-2-(aux*3)]+g_I[i-3-(aux*3)]+g_I[i-4-(aux*3)]+g_I[i+1-(aux*3)]+g_I[i+2-(aux*3)]+g_I[i+3-(aux*3)]+g_I[i+4-(aux*3)]+
            g_I[i+aux]+g_I[i-1+aux]+g_I[i-2+aux]+g_I[i-3+aux]+g_I[i-4+aux]+g_I[i+1+aux]+g_I[i+2+aux]+g_I[i+3+aux]+g_I[i+4+aux]+
            g_I[i+(aux*2)]+g_I[i-1+(aux*2)]+g_I[i-2+(aux*2)]+g_I[i-3+(aux*2)]+g_I[i-4+(aux*2)]+g_I[i+1+(aux*2)]+g_I[i+2+(aux*2)]+g_I[i+3+(aux*2)]+g_I[i+4+(aux*2)]+
            g_I[i+(aux*3)]+g_I[i-1+(aux*3)]+g_I[i-2+(aux*3)]+g_I[i-3+(aux*3)]+g_I[i-4+(aux*3)]+g_I[i+1+(aux*3)]+g_I[i+2+(aux*3)]+g_I[i+3+(aux*3)]+g_I[i+4+(aux*3)]+
 g_I[i+(aux*4)]+g_I[i-1+(aux*4)]+g_I[i-2+(aux*4)]+g_I[i-3+(aux*4)]+g_I[i-4+(aux*4)]+g_I[i+1+(aux*4)]+g_I[i+2+(aux*4)]+g_I[i+3+(aux*4)]+g_I[i+4+(aux*4)]+ g_I[i-(aux*4)]+g_I[i-1-(aux*4)]+g_I[i-2-(aux*4)]+g_I[i-3-(aux*4)]+g_I[i-4-(aux*4)]+g_I[i+1-(aux*4)]+g_I[i+2-(aux*4)]+g_I[i+3-(aux*4)]+g_I[i+4-(aux*4)]
  )/81;
	
	b_I[i] = (b_I[i]+b_I[i-1]+b_I[i-2]+b_I[i-3]+b_I[i-4]+b_I[i+1]+b_I[i+2]+b_I[i+3]+b_I[i+4]+
            b_I[i-aux]+b_I[i-1-aux]+b_I[i-2-aux]+b_I[i-3-aux]+b_I[i-4-aux]+b_I[i+1-aux]+b_I[i+2-aux]+b_I[i+3-aux]+b_I[i+4-aux]+
            b_I[i-(aux*2)]+b_I[i-1-(aux*2)]+b_I[i-2-(aux*2)]+b_I[i-3-(aux*2)]+b_I[i-4-(aux*2)]+b_I[i+1-(aux*2)]+b_I[i+2-(aux*2)]+b_I[i+3-(aux*2)]+b_I[i+4-(aux*2)]+
            b_I[i-(aux*3)]+b_I[i-1-(aux*3)]+b_I[i-2-(aux*3)]+b_I[i-3-(aux*3)]+b_I[i-4-(aux*3)]+b_I[i+1-(aux*3)]+b_I[i+2-(aux*3)]+b_I[i+3-(aux*3)]+b_I[i+4-(aux*3)]+
            b_I[i+aux]+b_I[i-1+aux]+b_I[i-2+aux]+b_I[i-3+aux]+b_I[i-4+aux]+b_I[i+1+aux]+b_I[i+2+aux]+b_I[i+3+aux]+b_I[i+4+aux]+
            b_I[i+(aux*2)]+b_I[i-1+(aux*2)]+b_I[i-2+(aux*2)]+b_I[i-3+(aux*2)]+b_I[i-4+(aux*2)]+b_I[i+1+(aux*2)]+b_I[i+2+(aux*2)]+b_I[i+3+(aux*2)]+b_I[i+4+(aux*2)]+
            b_I[i+(aux*3)]+b_I[i-1+(aux*3)]+b_I[i-2+(aux*3)]+b_I[i-3+(aux*3)]+b_I[i-4+(aux*3)]+b_I[i+1+(aux*3)]+b_I[i+2+(aux*3)]+b_I[i+3+(aux*3)]+b_I[i+4+(aux*3)]+
 b_I[i+(aux*4)]+b_I[i-1+(aux*4)]+b_I[i-2+(aux*4)]+b_I[i-3+(aux*4)]+b_I[i-4+(aux*4)]+b_I[i+1+(aux*4)]+b_I[i+2+(aux*4)]+b_I[i+3+(aux*4)]+b_I[i+4+(aux*4)]+ b_I[i-(aux*4)]+b_I[i-1-(aux*4)]+b_I[i-2-(aux*4)]+b_I[i-3-(aux*4)]+b_I[i-4-(aux*4)]+b_I[i+1-(aux*4)]+b_I[i+2-(aux*4)]+b_I[i+3-(aux*4)]+b_I[i+4-(aux*4)]
  )/81;

	

   	 }


	if(  nrows == 11){
		r_I[i] = ( r_I[i]+r_I[i-1]+r_I[i-2]+r_I[i-3]+r_I[i-4]+r_I[i+1]+r_I[i+2]+r_I[i+3]+r_I[i+4]+r_I[i+5]+
            r_I[i-aux]+r_I[i-1-aux]+r_I[i-2-aux]+r_I[i-3-aux]+r_I[i-4-aux]+r_I[i+1-aux]+r_I[i+2-aux]+r_I[i+3-aux]+ r_I[i+4-aux]+r_I[i+5-aux]+
     r_I[i-(aux*2)]+r_I[i-1-(aux*2)]+r_I[i-2-(aux*2)]+r_I[i-3-(aux*2)]+r_I[i-4-(aux*2)]+r_I[i+1-(aux*2)]+r_I[i+2-(aux*2)]+r_I[i+3-(aux*2)]+r_I[i+4-(aux*2)]+r_I[i+5-(aux*2)]+
            r_I[i-(aux*3)]+r_I[i-1-(aux*3)]+r_I[i-2-(aux*3)]+r_I[i-3-(aux*3)]+r_I[i-4-(aux*3)]+r_I[i+1-(aux*3)]+r_I[i+2-(aux*3)]+r_I[i+3-(aux*3)]+r_I[i+4-(aux*3)]+r_I[i+5-(aux*3)]+
            r_I[i+aux]+r_I[i-1+aux]+r_I[i-2+aux]+r_I[i-3+aux]+r_I[i-4+aux]+r_I[i+1+aux]+r_I[i+2+aux]+r_I[i+3+aux]+r_I[i+4+aux]+r_I[i+5+aux]+
            r_I[i+(aux*2)]+r_I[i-1+(aux*2)]+r_I[i-2+(aux*2)]+r_I[i-3+(aux*2)]+r_I[i-4+(aux*2)]+r_I[i+1+(aux*2)]+r_I[i+2+(aux*2)]+r_I[i+3+(aux*2)]+r_I[i+4+(aux*2)]+r_I[i+5+(aux*2)]+
            r_I[i+(aux*3)]+r_I[i-1+(aux*3)]+r_I[i-2+(aux*3)]+r_I[i-3+(aux*3)]+r_I[i-4+(aux*3)]+r_I[i+1+(aux*3)]+r_I[i+2+(aux*3)]+r_I[i+3+(aux*3)]+r_I[i+4+(aux*3)]+r_I[i+5+(aux*3)]+
 r_I[i+(aux*4)]+r_I[i-1+(aux*4)]+r_I[i-2+(aux*4)]+r_I[i-3+(aux*4)]+r_I[i-4+(aux*4)]+r_I[i-5+(aux*4)]+r_I[i+1+(aux*4)]+r_I[i+2+(aux*4)]+r_I[i+3+(aux*4)]+r_I[i+4+(aux*4)] +r_I[i+5+(aux*4)]+ + r_I[i-(aux*4)]+r_I[i-1-(aux*4)]+r_I[i-2-(aux*4)]+r_I[i-3-(aux*4)]+r_I[i-4-(aux*4)]+r_I[i-5-(aux*4)]+r_I[i+1-(aux*4)]+r_I[i+2-(aux*4)]+r_I[i+3-(aux*4)]+r_I[i+4-(aux*4)] +r_I[i+5-(aux*4)]   +
 r_I[i+(aux*5)]+r_I[i-1+(aux*5)]+r_I[i-2+(aux*5)]+r_I[i-3+(aux*5)]+r_I[i-4+(aux*5)]+r_I[i-5+(aux*5)]+r_I[i+1+(aux*5)]+r_I[i+2+(aux*5)]+r_I[i+3+(aux*5)]+r_I[i+4+(aux*5)] +r_I[i+5+(aux*5)]+
r_I[i-(aux*5)]+r_I[i-1-(aux*5)]+r_I[i-2-(aux*5)]+r_I[i-3-(aux*5)]+r_I[i-4-(aux*5)]+r_I[i-5-(aux*5)]+r_I[i+1-(aux*5)]+r_I[i+2-(aux*5)]+r_I[i+3-(aux*5)]+r_I[i+4-(aux*5)] +r_I[i+5-(aux*5)]
    )/121;

	g_I[i] = ( g_I[i]+g_I[i-1]+g_I[i-2]+g_I[i-3]+g_I[i-4]+g_I[i+1]+g_I[i+2]+g_I[i+3]+g_I[i+4]+g_I[i+5]+
            g_I[i-aux]+g_I[i-1-aux]+g_I[i-2-aux]+g_I[i-3-aux]+g_I[i-4-aux]+g_I[i+1-aux]+g_I[i+2-aux]+g_I[i+3-aux]+ g_I[i+4-aux]+g_I[i+5-aux]+
     g_I[i-(aux*2)]+g_I[i-1-(aux*2)]+g_I[i-2-(aux*2)]+g_I[i-3-(aux*2)]+g_I[i-4-(aux*2)]+g_I[i+1-(aux*2)]+g_I[i+2-(aux*2)]+g_I[i+3-(aux*2)]+g_I[i+4-(aux*2)]+g_I[i+5-(aux*2)]+
            g_I[i-(aux*3)]+g_I[i-1-(aux*3)]+g_I[i-2-(aux*3)]+g_I[i-3-(aux*3)]+g_I[i-4-(aux*3)]+g_I[i+1-(aux*3)]+g_I[i+2-(aux*3)]+g_I[i+3-(aux*3)]+g_I[i+4-(aux*3)]+g_I[i+5-(aux*3)]+
            g_I[i+aux]+g_I[i-1+aux]+g_I[i-2+aux]+g_I[i-3+aux]+g_I[i-4+aux]+g_I[i+1+aux]+g_I[i+2+aux]+g_I[i+3+aux]+g_I[i+4+aux]+g_I[i+5+aux]+
            g_I[i+(aux*2)]+g_I[i-1+(aux*2)]+g_I[i-2+(aux*2)]+g_I[i-3+(aux*2)]+g_I[i-4+(aux*2)]+g_I[i+1+(aux*2)]+g_I[i+2+(aux*2)]+g_I[i+3+(aux*2)]+g_I[i+4+(aux*2)]+g_I[i+5+(aux*2)]+
            g_I[i+(aux*3)]+g_I[i-1+(aux*3)]+g_I[i-2+(aux*3)]+g_I[i-3+(aux*3)]+g_I[i-4+(aux*3)]+g_I[i+1+(aux*3)]+g_I[i+2+(aux*3)]+g_I[i+3+(aux*3)]+g_I[i+4+(aux*3)]+g_I[i+5+(aux*3)]+
 g_I[i+(aux*4)]+g_I[i-1+(aux*4)]+g_I[i-2+(aux*4)]+g_I[i-3+(aux*4)]+g_I[i-4+(aux*4)]+g_I[i-5+(aux*4)]+g_I[i+1+(aux*4)]+g_I[i+2+(aux*4)]+g_I[i+3+(aux*4)]+g_I[i+4+(aux*4)] +g_I[i+5+(aux*4)]+ + g_I[i-(aux*4)]+g_I[i-1-(aux*4)]+g_I[i-2-(aux*4)]+g_I[i-3-(aux*4)]+g_I[i-4-(aux*4)]+g_I[i-5-(aux*4)]+g_I[i+1-(aux*4)]+g_I[i+2-(aux*4)]+g_I[i+3-(aux*4)]+g_I[i+4-(aux*4)] +g_I[i+5-(aux*4)]   +
 g_I[i+(aux*5)]+g_I[i-1+(aux*5)]+g_I[i-2+(aux*5)]+g_I[i-3+(aux*5)]+g_I[i-4+(aux*5)]+g_I[i-5+(aux*5)]+g_I[i+1+(aux*5)]+g_I[i+2+(aux*5)]+g_I[i+3+(aux*5)]+g_I[i+4+(aux*5)] +g_I[i+5+(aux*5)]+
g_I[i-(aux*5)]+g_I[i-1-(aux*5)]+g_I[i-2-(aux*5)]+g_I[i-3-(aux*5)]+g_I[i-4-(aux*5)]+g_I[i-5-(aux*5)]+g_I[i+1-(aux*5)]+g_I[i+2-(aux*5)]+g_I[i+3-(aux*5)]+g_I[i+4-(aux*5)] +g_I[i+5-(aux*5)]
    )/121;
	
b_I[i] = ( b_I[i]+b_I[i-1]+b_I[i-2]+b_I[i-3]+b_I[i-4]+b_I[i+1]+b_I[i+2]+b_I[i+3]+b_I[i+4]+b_I[i+5]+
            b_I[i-aux]+b_I[i-1-aux]+b_I[i-2-aux]+b_I[i-3-aux]+b_I[i-4-aux]+b_I[i+1-aux]+b_I[i+2-aux]+b_I[i+3-aux]+ b_I[i+4-aux]+b_I[i+5-aux]+
     b_I[i-(aux*2)]+b_I[i-1-(aux*2)]+b_I[i-2-(aux*2)]+b_I[i-3-(aux*2)]+b_I[i-4-(aux*2)]+b_I[i+1-(aux*2)]+b_I[i+2-(aux*2)]+b_I[i+3-(aux*2)]+b_I[i+4-(aux*2)]+b_I[i+5-(aux*2)]+
            b_I[i-(aux*3)]+b_I[i-1-(aux*3)]+b_I[i-2-(aux*3)]+b_I[i-3-(aux*3)]+b_I[i-4-(aux*3)]+b_I[i+1-(aux*3)]+b_I[i+2-(aux*3)]+b_I[i+3-(aux*3)]+b_I[i+4-(aux*3)]+b_I[i+5-(aux*3)]+
            b_I[i+aux]+b_I[i-1+aux]+b_I[i-2+aux]+b_I[i-3+aux]+b_I[i-4+aux]+b_I[i+1+aux]+b_I[i+2+aux]+b_I[i+3+aux]+b_I[i+4+aux]+b_I[i+5+aux]+
            b_I[i+(aux*2)]+b_I[i-1+(aux*2)]+b_I[i-2+(aux*2)]+b_I[i-3+(aux*2)]+b_I[i-4+(aux*2)]+b_I[i+1+(aux*2)]+b_I[i+2+(aux*2)]+b_I[i+3+(aux*2)]+b_I[i+4+(aux*2)]+b_I[i+5+(aux*2)]+
            b_I[i+(aux*3)]+b_I[i-1+(aux*3)]+b_I[i-2+(aux*3)]+b_I[i-3+(aux*3)]+b_I[i-4+(aux*3)]+b_I[i+1+(aux*3)]+b_I[i+2+(aux*3)]+b_I[i+3+(aux*3)]+b_I[i+4+(aux*3)]+b_I[i+5+(aux*3)]+
 b_I[i+(aux*4)]+b_I[i-1+(aux*4)]+b_I[i-2+(aux*4)]+b_I[i-3+(aux*4)]+b_I[i-4+(aux*4)]+b_I[i-5+(aux*4)]+b_I[i+1+(aux*4)]+b_I[i+2+(aux*4)]+b_I[i+3+(aux*4)]+b_I[i+4+(aux*4)] +b_I[i+5+(aux*4)]+ + b_I[i-(aux*4)]+b_I[i-1-(aux*4)]+b_I[i-2-(aux*4)]+b_I[i-3-(aux*4)]+b_I[i-4-(aux*4)]+b_I[i-5-(aux*4)]+b_I[i+1-(aux*4)]+b_I[i+2-(aux*4)]+b_I[i+3-(aux*4)]+b_I[i+4-(aux*4)] +b_I[i+5-(aux*4)]   +
 b_I[i+(aux*5)]+b_I[i-1+(aux*5)]+b_I[i-2+(aux*5)]+b_I[i-3+(aux*5)]+b_I[i-4+(aux*5)]+b_I[i-5+(aux*5)]+b_I[i+1+(aux*5)]+b_I[i+2+(aux*5)]+b_I[i+3+(aux*5)]+b_I[i+4+(aux*5)] +b_I[i+5+(aux*5)]+
b_I[i-(aux*5)]+b_I[i-1-(aux*5)]+b_I[i-2-(aux*5)]+b_I[i-3-(aux*5)]+b_I[i-4-(aux*5)]+b_I[i-5-(aux*5)]+b_I[i+1-(aux*5)]+b_I[i+2-(aux*5)]+b_I[i+3-(aux*5)]+b_I[i+4-(aux*5)] +b_I[i+5-(aux*5)]
    )/121;


   	 }

	if(  nrows == 13){
		r_I[i] = ( r_I[i]+r_I[i-1]+r_I[i-2]+r_I[i-3]+r_I[i-4]+r_I[i+1]+r_I[i+2]+r_I[i+3]+r_I[i+4]+r_I[i+5]+r_I[i+6]+
            r_I[i-aux]+r_I[i-1-aux]+r_I[i-2-aux]+r_I[i-3-aux]+r_I[i-4-aux]+r_I[i+1-aux]+r_I[i+2-aux]+r_I[i+3-aux]+ r_I[i+4-aux]+r_I[i+5-aux]+r_I[i+6-aux]+
     r_I[i-(aux*2)]+r_I[i-1-(aux*2)]+r_I[i-2-(aux*2)]+r_I[i-3-(aux*2)]+r_I[i-4-(aux*2)]+r_I[i+1-(aux*2)]+r_I[i+2-(aux*2)]+r_I[i+3-(aux*2)]+r_I[i+4-(aux*2)]+r_I[i+5-(aux*2)]+r_I[i+6-(aux*2)]+
            r_I[i-(aux*3)]+r_I[i-1-(aux*3)]+r_I[i-2-(aux*3)]+r_I[i-3-(aux*3)]+r_I[i-4-(aux*3)]+r_I[i+1-(aux*3)]+r_I[i+2-(aux*3)]+r_I[i+3-(aux*3)]+r_I[i+4-(aux*3)]+r_I[i+5-(aux*3)]+r_I[i+6-(aux*3)]+
            r_I[i+aux]+r_I[i-1+aux]+r_I[i-2+aux]+r_I[i-3+aux]+r_I[i-4+aux]+r_I[i+1+aux]+r_I[i+2+aux]+r_I[i+3+aux]+r_I[i+4+aux]+r_I[i+5+aux]+r_I[i+6+aux]+
            r_I[i+(aux*2)]+r_I[i-1+(aux*2)]+r_I[i-2+(aux*2)]+r_I[i-3+(aux*2)]+r_I[i-4+(aux*2)]+r_I[i+1+(aux*2)]+r_I[i+2+(aux*2)]+r_I[i+3+(aux*2)]+r_I[i+4+(aux*2)]+r_I[i+5+(aux*2)]+r_I[i+6+(aux*2)]+
            r_I[i+(aux*3)]+r_I[i-1+(aux*3)]+r_I[i-2+(aux*3)]+r_I[i-3+(aux*3)]+r_I[i-4+(aux*3)]+r_I[i+1+(aux*3)]+r_I[i+2+(aux*3)]+r_I[i+3+(aux*3)]+r_I[i+4+(aux*3)]+r_I[i+5+(aux*3)]+r_I[i+6+(aux*3)]+
 r_I[i+(aux*4)]+r_I[i-1+(aux*4)]+r_I[i-2+(aux*4)]+r_I[i-3+(aux*4)]+r_I[i-4+(aux*4)]+r_I[i-5+(aux*4)]+r_I[i-6+(aux*4)]
+r_I[i+1+(aux*4)]+r_I[i+2+(aux*4)]+r_I[i+3+(aux*4)]+r_I[i+4+(aux*4)] +r_I[i+5+(aux*4)] +r_I[i+6+(aux*4)]
+ r_I[i-(aux*4)]+r_I[i-1-(aux*4)]+r_I[i-2-(aux*4)]+r_I[i-3-(aux*4)]+r_I[i-4-(aux*4)]+r_I[i-5-(aux*4)]+r_I[i-6-(aux*4)]
+r_I[i+1-(aux*4)]+r_I[i+2-(aux*4)]+r_I[i+3-(aux*4)]+r_I[i+4-(aux*4)] +r_I[i+5-(aux*4)] +r_I[i+6-(aux*4)]  +
 r_I[i+(aux*5)]+r_I[i-1+(aux*5)]+r_I[i-2+(aux*5)]+r_I[i-3+(aux*5)]+r_I[i-4+(aux*5)]+r_I[i-5+(aux*5)]+r_I[i-6+(aux*5)]
+r_I[i+1+(aux*5)]+r_I[i+2+(aux*5)]+r_I[i+3+(aux*5)]+r_I[i+4+(aux*5)] +r_I[i+5+(aux*5)]+r_I[i+6+(aux*5)]
+ r_I[i-(aux*5)]+r_I[i-1-(aux*5)]+r_I[i-2-(aux*5)]+r_I[i-3-(aux*5)]+r_I[i-4-(aux*5)]+r_I[i-5-(aux*5)]+r_I[i+1-(aux*5)]+r_I[i+2-(aux*5)]+r_I[i+3-(aux*5)]+r_I[i+4-(aux*5)] +r_I[i+5-(aux*5)]+r_I[i+6-(aux*5)]+
r_I[i+(aux*5)]+r_I[i-1+(aux*5)]+r_I[i-2+(aux*5)]+r_I[i-3+(aux*5)]+r_I[i-4+(aux*5)]+r_I[i-5+(aux*5)]+r_I[i-6+(aux*5)]
+r_I[i+1+(aux*6)]+r_I[i+2+(aux*6)]+r_I[i+3+(aux*6)]+r_I[i+4+(aux*6)] +r_I[i+5+(aux*6)]+r_I[i+6+(aux*6)]
+ r_I[i-(aux*6)]+r_I[i-1-(aux*6)]+r_I[i-2-(aux*6)]+r_I[i-3-(aux*6)]+r_I[i-4-(aux*6)]+r_I[i-5-(aux*6)]+r_I[i+1-(aux*6)]+r_I[i+2-(aux*6)]+r_I[i+3-(aux*6)]+r_I[i+4-(aux*6)] +r_I[i+5-(aux*6)]+r_I[i+6-(aux*6)]
    )/169;

	g_I[i] = ( g_I[i]+g_I[i-1]+g_I[i-2]+g_I[i-3]+g_I[i-4]+g_I[i+1]+g_I[i+2]+g_I[i+3]+g_I[i+4]+g_I[i+5]+g_I[i+6]+
            g_I[i-aux]+g_I[i-1-aux]+g_I[i-2-aux]+g_I[i-3-aux]+g_I[i-4-aux]+g_I[i+1-aux]+g_I[i+2-aux]+g_I[i+3-aux]+ g_I[i+4-aux]+g_I[i+5-aux]+g_I[i+6-aux]+
     g_I[i-(aux*2)]+g_I[i-1-(aux*2)]+g_I[i-2-(aux*2)]+g_I[i-3-(aux*2)]+g_I[i-4-(aux*2)]+g_I[i+1-(aux*2)]+g_I[i+2-(aux*2)]+g_I[i+3-(aux*2)]+g_I[i+4-(aux*2)]+g_I[i+5-(aux*2)]+g_I[i+6-(aux*2)]+
            g_I[i-(aux*3)]+g_I[i-1-(aux*3)]+g_I[i-2-(aux*3)]+g_I[i-3-(aux*3)]+g_I[i-4-(aux*3)]+g_I[i+1-(aux*3)]+g_I[i+2-(aux*3)]+g_I[i+3-(aux*3)]+g_I[i+4-(aux*3)]+g_I[i+5-(aux*3)]+g_I[i+6-(aux*3)]+
            g_I[i+aux]+g_I[i-1+aux]+g_I[i-2+aux]+g_I[i-3+aux]+g_I[i-4+aux]+g_I[i+1+aux]+g_I[i+2+aux]+g_I[i+3+aux]+g_I[i+4+aux]+g_I[i+5+aux]+g_I[i+6+aux]+
            g_I[i+(aux*2)]+g_I[i-1+(aux*2)]+g_I[i-2+(aux*2)]+g_I[i-3+(aux*2)]+g_I[i-4+(aux*2)]+g_I[i+1+(aux*2)]+g_I[i+2+(aux*2)]+g_I[i+3+(aux*2)]+g_I[i+4+(aux*2)]+g_I[i+5+(aux*2)]+g_I[i+6+(aux*2)]+
            g_I[i+(aux*3)]+g_I[i-1+(aux*3)]+g_I[i-2+(aux*3)]+g_I[i-3+(aux*3)]+g_I[i-4+(aux*3)]+g_I[i+1+(aux*3)]+g_I[i+2+(aux*3)]+g_I[i+3+(aux*3)]+g_I[i+4+(aux*3)]+g_I[i+5+(aux*3)]+g_I[i+6+(aux*3)]+
 g_I[i+(aux*4)]+g_I[i-1+(aux*4)]+g_I[i-2+(aux*4)]+g_I[i-3+(aux*4)]+g_I[i-4+(aux*4)]+g_I[i-5+(aux*4)]+g_I[i-6+(aux*4)]
+g_I[i+1+(aux*4)]+g_I[i+2+(aux*4)]+g_I[i+3+(aux*4)]+g_I[i+4+(aux*4)] +g_I[i+5+(aux*4)] +g_I[i+6+(aux*4)]
+ g_I[i-(aux*4)]+g_I[i-1-(aux*4)]+g_I[i-2-(aux*4)]+g_I[i-3-(aux*4)]+g_I[i-4-(aux*4)]+g_I[i-5-(aux*4)]+g_I[i-6-(aux*4)]
+g_I[i+1-(aux*4)]+g_I[i+2-(aux*4)]+g_I[i+3-(aux*4)]+g_I[i+4-(aux*4)] +g_I[i+5-(aux*4)] +g_I[i+6-(aux*4)]  +
 g_I[i+(aux*5)]+g_I[i-1+(aux*5)]+g_I[i-2+(aux*5)]+g_I[i-3+(aux*5)]+g_I[i-4+(aux*5)]+g_I[i-5+(aux*5)]+g_I[i-6+(aux*5)]
+g_I[i+1+(aux*5)]+g_I[i+2+(aux*5)]+g_I[i+3+(aux*5)]+g_I[i+4+(aux*5)] +g_I[i+5+(aux*5)]+g_I[i+6+(aux*5)]
+ g_I[i-(aux*5)]+g_I[i-1-(aux*5)]+g_I[i-2-(aux*5)]+g_I[i-3-(aux*5)]+g_I[i-4-(aux*5)]+g_I[i-5-(aux*5)]+g_I[i+1-(aux*5)]+g_I[i+2-(aux*5)]+g_I[i+3-(aux*5)]+g_I[i+4-(aux*5)] +g_I[i+5-(aux*5)]+g_I[i+6-(aux*5)]+
g_I[i+(aux*5)]+g_I[i-1+(aux*5)]+g_I[i-2+(aux*5)]+g_I[i-3+(aux*5)]+g_I[i-4+(aux*5)]+g_I[i-5+(aux*5)]+g_I[i-6+(aux*5)]
+g_I[i+1+(aux*6)]+g_I[i+2+(aux*6)]+g_I[i+3+(aux*6)]+g_I[i+4+(aux*6)] +g_I[i+5+(aux*6)]+g_I[i+6+(aux*6)]
+ g_I[i-(aux*6)]+g_I[i-1-(aux*6)]+g_I[i-2-(aux*6)]+g_I[i-3-(aux*6)]+g_I[i-4-(aux*6)]+g_I[i-5-(aux*6)]+g_I[i+1-(aux*6)]+g_I[i+2-(aux*6)]+g_I[i+3-(aux*6)]+g_I[i+4-(aux*6)] +g_I[i+5-(aux*6)]+g_I[i+6-(aux*6)]
    )/169;


	b_I[i] = ( b_I[i]+b_I[i-1]+b_I[i-2]+b_I[i-3]+b_I[i-4]+b_I[i+1]+b_I[i+2]+b_I[i+3]+b_I[i+4]+b_I[i+5]+b_I[i+6]+
            b_I[i-aux]+b_I[i-1-aux]+b_I[i-2-aux]+b_I[i-3-aux]+b_I[i-4-aux]+b_I[i+1-aux]+b_I[i+2-aux]+b_I[i+3-aux]+ b_I[i+4-aux]+b_I[i+5-aux]+b_I[i+6-aux]+
     b_I[i-(aux*2)]+b_I[i-1-(aux*2)]+b_I[i-2-(aux*2)]+b_I[i-3-(aux*2)]+b_I[i-4-(aux*2)]+b_I[i+1-(aux*2)]+b_I[i+2-(aux*2)]+b_I[i+3-(aux*2)]+b_I[i+4-(aux*2)]+b_I[i+5-(aux*2)]+b_I[i+6-(aux*2)]+
            b_I[i-(aux*3)]+b_I[i-1-(aux*3)]+b_I[i-2-(aux*3)]+b_I[i-3-(aux*3)]+b_I[i-4-(aux*3)]+b_I[i+1-(aux*3)]+b_I[i+2-(aux*3)]+b_I[i+3-(aux*3)]+b_I[i+4-(aux*3)]+b_I[i+5-(aux*3)]+b_I[i+6-(aux*3)]+
            b_I[i+aux]+b_I[i-1+aux]+b_I[i-2+aux]+b_I[i-3+aux]+b_I[i-4+aux]+b_I[i+1+aux]+b_I[i+2+aux]+b_I[i+3+aux]+b_I[i+4+aux]+b_I[i+5+aux]+b_I[i+6+aux]+
            b_I[i+(aux*2)]+b_I[i-1+(aux*2)]+b_I[i-2+(aux*2)]+b_I[i-3+(aux*2)]+b_I[i-4+(aux*2)]+b_I[i+1+(aux*2)]+b_I[i+2+(aux*2)]+b_I[i+3+(aux*2)]+b_I[i+4+(aux*2)]+b_I[i+5+(aux*2)]+b_I[i+6+(aux*2)]+
            b_I[i+(aux*3)]+b_I[i-1+(aux*3)]+b_I[i-2+(aux*3)]+b_I[i-3+(aux*3)]+b_I[i-4+(aux*3)]+b_I[i+1+(aux*3)]+b_I[i+2+(aux*3)]+b_I[i+3+(aux*3)]+b_I[i+4+(aux*3)]+b_I[i+5+(aux*3)]+b_I[i+6+(aux*3)]+
 b_I[i+(aux*4)]+b_I[i-1+(aux*4)]+b_I[i-2+(aux*4)]+b_I[i-3+(aux*4)]+b_I[i-4+(aux*4)]+b_I[i-5+(aux*4)]+b_I[i-6+(aux*4)]
+b_I[i+1+(aux*4)]+b_I[i+2+(aux*4)]+b_I[i+3+(aux*4)]+b_I[i+4+(aux*4)] +b_I[i+5+(aux*4)] +b_I[i+6+(aux*4)]
+ b_I[i-(aux*4)]+b_I[i-1-(aux*4)]+b_I[i-2-(aux*4)]+b_I[i-3-(aux*4)]+b_I[i-4-(aux*4)]+b_I[i-5-(aux*4)]+b_I[i-6-(aux*4)]
+b_I[i+1-(aux*4)]+b_I[i+2-(aux*4)]+b_I[i+3-(aux*4)]+b_I[i+4-(aux*4)] +b_I[i+5-(aux*4)] +b_I[i+6-(aux*4)]  +
 b_I[i+(aux*5)]+b_I[i-1+(aux*5)]+b_I[i-2+(aux*5)]+b_I[i-3+(aux*5)]+b_I[i-4+(aux*5)]+b_I[i-5+(aux*5)]+b_I[i-6+(aux*5)]
+b_I[i+1+(aux*5)]+b_I[i+2+(aux*5)]+b_I[i+3+(aux*5)]+b_I[i+4+(aux*5)] +b_I[i+5+(aux*5)]+b_I[i+6+(aux*5)]
+ b_I[i-(aux*5)]+b_I[i-1-(aux*5)]+b_I[i-2-(aux*5)]+b_I[i-3-(aux*5)]+b_I[i-4-(aux*5)]+b_I[i-5-(aux*5)]+b_I[i+1-(aux*5)]+b_I[i+2-(aux*5)]+b_I[i+3-(aux*5)]+b_I[i+4-(aux*5)] +b_I[i+5-(aux*5)]+b_I[i+6-(aux*5)]+
b_I[i+(aux*5)]+b_I[i-1+(aux*5)]+b_I[i-2+(aux*5)]+b_I[i-3+(aux*5)]+b_I[i-4+(aux*5)]+b_I[i-5+(aux*5)]+b_I[i-6+(aux*5)]
+b_I[i+1+(aux*6)]+b_I[i+2+(aux*6)]+b_I[i+3+(aux*6)]+b_I[i+4+(aux*6)] +b_I[i+5+(aux*6)]+b_I[i+6+(aux*6)]
+ b_I[i-(aux*6)]+b_I[i-1-(aux*6)]+b_I[i-2-(aux*6)]+b_I[i-3-(aux*6)]+b_I[i-4-(aux*6)]+b_I[i-5-(aux*6)]+b_I[i+1-(aux*6)]+b_I[i+2-(aux*6)]+b_I[i+3-(aux*6)]+b_I[i+4-(aux*6)] +b_I[i+5-(aux*6)]+b_I[i+6-(aux*6)]
    )/169;



   	 }


	if(  nrows == 15){
		r_I[i] = ( r_I[i]+r_I[i-1]+r_I[i-2]+r_I[i-3]+r_I[i-4]+r_I[i+1]+r_I[i+2]+r_I[i+3]+r_I[i+4]+r_I[i+5]+r_I[i+6]+r_I[i+7]+
            r_I[i-aux]+r_I[i-1-aux]+r_I[i-2-aux]+r_I[i-3-aux]+r_I[i-4-aux]+r_I[i+1-aux]+r_I[i+2-aux]+r_I[i+3-aux]+ r_I[i+4-aux]+r_I[i+5-aux]+r_I[i+6-aux]+r_I[i+7-aux]+
     r_I[i-(aux*2)]+r_I[i-1-(aux*2)]+r_I[i-2-(aux*2)]+r_I[i-3-(aux*2)]+r_I[i-4-(aux*2)]+r_I[i+1-(aux*2)]+r_I[i+2-(aux*2)]+r_I[i+3-(aux*2)]+r_I[i+4-(aux*2)]+r_I[i+5-(aux*2)]+r_I[i+6-(aux*2)]+r_I[i+7-(aux*2)]+
            r_I[i-(aux*3)]+r_I[i-1-(aux*3)]+r_I[i-2-(aux*3)]+r_I[i-3-(aux*3)]+r_I[i-4-(aux*3)]+r_I[i+1-(aux*3)]+r_I[i+2-(aux*3)]+r_I[i+3-(aux*3)]+r_I[i+4-(aux*3)]+r_I[i+5-(aux*3)]+r_I[i+6-(aux*3)]+r_I[i+7-(aux*3)]+
            r_I[i+aux]+r_I[i-1+aux]+r_I[i-2+aux]+r_I[i-3+aux]+r_I[i-4+aux]+r_I[i+1+aux]+r_I[i+2+aux]+r_I[i+3+aux]+r_I[i+4+aux]+r_I[i+5+aux]+r_I[i+6+aux]+r_I[i+7+aux]+
            r_I[i+(aux*2)]+r_I[i-1+(aux*2)]+r_I[i-2+(aux*2)]+r_I[i-3+(aux*2)]+r_I[i-4+(aux*2)]+r_I[i+1+(aux*2)]+r_I[i+2+(aux*2)]+r_I[i+3+(aux*2)]+r_I[i+4+(aux*2)]+r_I[i+5+(aux*2)]+r_I[i+6+(aux*2)]+r_I[i+7+(aux*2)]+
            r_I[i+(aux*3)]+r_I[i-1+(aux*3)]+r_I[i-2+(aux*3)]+r_I[i-3+(aux*3)]+r_I[i-4+(aux*3)]+r_I[i+1+(aux*3)]+r_I[i+2+(aux*3)]+r_I[i+3+(aux*3)]+r_I[i+4+(aux*3)]+r_I[i+5+(aux*3)]+r_I[i+6+(aux*3)]+r_I[i+7+(aux*3)]+
 r_I[i+(aux*4)]+r_I[i-1+(aux*4)]+r_I[i-2+(aux*4)]+r_I[i-3+(aux*4)]+r_I[i-4+(aux*4)]+r_I[i-5+(aux*4)]+r_I[i-6+(aux*4)]+r_I[i-7+(aux*4)]
+r_I[i+1+(aux*4)]+r_I[i+2+(aux*4)]+r_I[i+3+(aux*4)]+r_I[i+4+(aux*4)] +r_I[i+5+(aux*4)] +r_I[i+6+(aux*4)]
+ r_I[i-(aux*4)]+r_I[i-1-(aux*4)]+r_I[i-2-(aux*4)]+r_I[i-3-(aux*4)]+r_I[i-4-(aux*4)]+r_I[i-5-(aux*4)]+r_I[i-6-(aux*4)]+r_I[i-7-(aux*4)]
+r_I[i+1-(aux*4)]+r_I[i+2-(aux*4)]+r_I[i+3-(aux*4)]+r_I[i+4-(aux*4)] +r_I[i+5-(aux*4)] +r_I[i+6-(aux*4)] +r_I[i+7-(aux*4)]  +
 r_I[i+(aux*5)]+r_I[i-1+(aux*5)]+r_I[i-2+(aux*5)]+r_I[i-3+(aux*5)]+r_I[i-4+(aux*5)]+r_I[i-5+(aux*5)]+r_I[i-6+(aux*5)]+r_I[i-7+(aux*5)]
+r_I[i+1+(aux*5)]+r_I[i+2+(aux*5)]+r_I[i+3+(aux*5)]+r_I[i+4+(aux*5)] +r_I[i+5+(aux*5)]+r_I[i+6+(aux*5)]+r_I[i+7+(aux*5)]
+ r_I[i-(aux*5)]+r_I[i-1-(aux*5)]+r_I[i-2-(aux*5)]+r_I[i-3-(aux*5)]+r_I[i-4-(aux*5)]+r_I[i-5-(aux*5)]+r_I[i+1-(aux*5)]+r_I[i+2-(aux*5)]+r_I[i+3-(aux*5)]+r_I[i+4-(aux*5)] +r_I[i+5-(aux*5)]+r_I[i+6-(aux*5)]+r_I[i+7-(aux*5)]+
r_I[i+(aux*5)]+r_I[i-1+(aux*5)]+r_I[i-2+(aux*5)]+r_I[i-3+(aux*5)]+r_I[i-4+(aux*5)]+r_I[i-5+(aux*5)]+r_I[i-6+(aux*5)]+r_I[i-7+(aux*5)]
+r_I[i+1+(aux*6)]+r_I[i+2+(aux*6)]+r_I[i+3+(aux*6)]+r_I[i+4+(aux*6)] +r_I[i+5+(aux*6)]+r_I[i+6+(aux*6)]+r_I[i+7+(aux*6)]
+ r_I[i-(aux*6)]+r_I[i-1-(aux*6)]+r_I[i-2-(aux*6)]+r_I[i-3-(aux*6)]+r_I[i-4-(aux*6)]+r_I[i-5-(aux*6)]+r_I[i+1-(aux*6)]+r_I[i+2-(aux*6)]+r_I[i+3-(aux*6)]+r_I[i+4-(aux*6)] +r_I[i+5-(aux*6)]+r_I[i+6-(aux*6)]+r_I[i+7-(aux*6)]

+r_I[i+1+(aux*7)]+r_I[i+2+(aux*7)]+r_I[i+3+(aux*7)]+r_I[i+4+(aux*7)] +r_I[i+5+(aux*7)]+r_I[i+6+(aux*7)]+r_I[i+7+(aux*7)]
+ r_I[i-(aux*7)]+r_I[i-1-(aux*7)]+r_I[i-2-(aux*7)]+r_I[i-3-(aux*7)]+r_I[i-4-(aux*7)]+r_I[i-5-(aux*7)]+r_I[i+1-(aux*7)]+r_I[i+2-(aux*7)]+r_I[i+3-(aux*7)]+r_I[i+4-(aux*7)] +r_I[i+5-(aux*7)]+r_I[i+6-(aux*7)]+r_I[i+7-(aux*7)]
    )/225;

	b_I[i] = ( b_I[i]+b_I[i-1]+b_I[i-2]+b_I[i-3]+b_I[i-4]+b_I[i+1]+b_I[i+2]+b_I[i+3]+b_I[i+4]+b_I[i+5]+b_I[i+6]+b_I[i+7]+
            b_I[i-aux]+b_I[i-1-aux]+b_I[i-2-aux]+b_I[i-3-aux]+b_I[i-4-aux]+b_I[i+1-aux]+b_I[i+2-aux]+b_I[i+3-aux]+ b_I[i+4-aux]+b_I[i+5-aux]+b_I[i+6-aux]+b_I[i+7-aux]+
     b_I[i-(aux*2)]+b_I[i-1-(aux*2)]+b_I[i-2-(aux*2)]+b_I[i-3-(aux*2)]+b_I[i-4-(aux*2)]+b_I[i+1-(aux*2)]+b_I[i+2-(aux*2)]+b_I[i+3-(aux*2)]+b_I[i+4-(aux*2)]+b_I[i+5-(aux*2)]+b_I[i+6-(aux*2)]+b_I[i+7-(aux*2)]+
            b_I[i-(aux*3)]+b_I[i-1-(aux*3)]+b_I[i-2-(aux*3)]+b_I[i-3-(aux*3)]+b_I[i-4-(aux*3)]+b_I[i+1-(aux*3)]+b_I[i+2-(aux*3)]+b_I[i+3-(aux*3)]+b_I[i+4-(aux*3)]+b_I[i+5-(aux*3)]+b_I[i+6-(aux*3)]+b_I[i+7-(aux*3)]+
            b_I[i+aux]+b_I[i-1+aux]+b_I[i-2+aux]+b_I[i-3+aux]+b_I[i-4+aux]+b_I[i+1+aux]+b_I[i+2+aux]+b_I[i+3+aux]+b_I[i+4+aux]+b_I[i+5+aux]+b_I[i+6+aux]+b_I[i+7+aux]+
            b_I[i+(aux*2)]+b_I[i-1+(aux*2)]+b_I[i-2+(aux*2)]+b_I[i-3+(aux*2)]+b_I[i-4+(aux*2)]+b_I[i+1+(aux*2)]+b_I[i+2+(aux*2)]+b_I[i+3+(aux*2)]+b_I[i+4+(aux*2)]+b_I[i+5+(aux*2)]+b_I[i+6+(aux*2)]+b_I[i+7+(aux*2)]+
            b_I[i+(aux*3)]+b_I[i-1+(aux*3)]+b_I[i-2+(aux*3)]+b_I[i-3+(aux*3)]+b_I[i-4+(aux*3)]+b_I[i+1+(aux*3)]+b_I[i+2+(aux*3)]+b_I[i+3+(aux*3)]+b_I[i+4+(aux*3)]+b_I[i+5+(aux*3)]+b_I[i+6+(aux*3)]+b_I[i+7+(aux*3)]+
 b_I[i+(aux*4)]+b_I[i-1+(aux*4)]+b_I[i-2+(aux*4)]+b_I[i-3+(aux*4)]+b_I[i-4+(aux*4)]+b_I[i-5+(aux*4)]+b_I[i-6+(aux*4)]+b_I[i-7+(aux*4)]
+b_I[i+1+(aux*4)]+b_I[i+2+(aux*4)]+b_I[i+3+(aux*4)]+b_I[i+4+(aux*4)] +b_I[i+5+(aux*4)] +b_I[i+6+(aux*4)]
+ b_I[i-(aux*4)]+b_I[i-1-(aux*4)]+b_I[i-2-(aux*4)]+b_I[i-3-(aux*4)]+b_I[i-4-(aux*4)]+b_I[i-5-(aux*4)]+b_I[i-6-(aux*4)]+b_I[i-7-(aux*4)]
+b_I[i+1-(aux*4)]+b_I[i+2-(aux*4)]+b_I[i+3-(aux*4)]+b_I[i+4-(aux*4)] +b_I[i+5-(aux*4)] +b_I[i+6-(aux*4)] +b_I[i+7-(aux*4)]  +
 b_I[i+(aux*5)]+b_I[i-1+(aux*5)]+b_I[i-2+(aux*5)]+b_I[i-3+(aux*5)]+b_I[i-4+(aux*5)]+b_I[i-5+(aux*5)]+b_I[i-6+(aux*5)]+b_I[i-7+(aux*5)]
+b_I[i+1+(aux*5)]+b_I[i+2+(aux*5)]+b_I[i+3+(aux*5)]+b_I[i+4+(aux*5)] +b_I[i+5+(aux*5)]+b_I[i+6+(aux*5)]+b_I[i+7+(aux*5)]
+ b_I[i-(aux*5)]+b_I[i-1-(aux*5)]+b_I[i-2-(aux*5)]+b_I[i-3-(aux*5)]+b_I[i-4-(aux*5)]+b_I[i-5-(aux*5)]+b_I[i+1-(aux*5)]+b_I[i+2-(aux*5)]+b_I[i+3-(aux*5)]+b_I[i+4-(aux*5)] +b_I[i+5-(aux*5)]+b_I[i+6-(aux*5)]+b_I[i+7-(aux*5)]+
b_I[i+(aux*5)]+b_I[i-1+(aux*5)]+b_I[i-2+(aux*5)]+b_I[i-3+(aux*5)]+b_I[i-4+(aux*5)]+b_I[i-5+(aux*5)]+b_I[i-6+(aux*5)]+b_I[i-7+(aux*5)]
+b_I[i+1+(aux*6)]+b_I[i+2+(aux*6)]+b_I[i+3+(aux*6)]+b_I[i+4+(aux*6)] +b_I[i+5+(aux*6)]+b_I[i+6+(aux*6)]+b_I[i+7+(aux*6)]
+ b_I[i-(aux*6)]+b_I[i-1-(aux*6)]+b_I[i-2-(aux*6)]+b_I[i-3-(aux*6)]+b_I[i-4-(aux*6)]+b_I[i-5-(aux*6)]+b_I[i+1-(aux*6)]+b_I[i+2-(aux*6)]+b_I[i+3-(aux*6)]+b_I[i+4-(aux*6)] +b_I[i+5-(aux*6)]+b_I[i+6-(aux*6)]+b_I[i+7-(aux*6)]

+b_I[i+1+(aux*7)]+b_I[i+2+(aux*7)]+b_I[i+3+(aux*7)]+b_I[i+4+(aux*7)] +b_I[i+5+(aux*7)]+b_I[i+6+(aux*7)]+b_I[i+7+(aux*7)]
+ b_I[i-(aux*7)]+b_I[i-1-(aux*7)]+b_I[i-2-(aux*7)]+b_I[i-3-(aux*7)]+b_I[i-4-(aux*7)]+b_I[i-5-(aux*7)]+b_I[i+1-(aux*7)]+b_I[i+2-(aux*7)]+b_I[i+3-(aux*7)]+b_I[i+4-(aux*7)] +b_I[i+5-(aux*7)]+b_I[i+6-(aux*7)]+b_I[i+7-(aux*7)]
    )/225;


	g_I[i] = ( g_I[i]+g_I[i-1]+g_I[i-2]+g_I[i-3]+g_I[i-4]+g_I[i+1]+g_I[i+2]+g_I[i+3]+g_I[i+4]+g_I[i+5]+g_I[i+6]+g_I[i+7]+
            g_I[i-aux]+g_I[i-1-aux]+g_I[i-2-aux]+g_I[i-3-aux]+g_I[i-4-aux]+g_I[i+1-aux]+g_I[i+2-aux]+g_I[i+3-aux]+ g_I[i+4-aux]+g_I[i+5-aux]+g_I[i+6-aux]+g_I[i+7-aux]+
     g_I[i-(aux*2)]+g_I[i-1-(aux*2)]+g_I[i-2-(aux*2)]+g_I[i-3-(aux*2)]+g_I[i-4-(aux*2)]+g_I[i+1-(aux*2)]+g_I[i+2-(aux*2)]+g_I[i+3-(aux*2)]+g_I[i+4-(aux*2)]+g_I[i+5-(aux*2)]+g_I[i+6-(aux*2)]+g_I[i+7-(aux*2)]+
            g_I[i-(aux*3)]+g_I[i-1-(aux*3)]+g_I[i-2-(aux*3)]+g_I[i-3-(aux*3)]+g_I[i-4-(aux*3)]+g_I[i+1-(aux*3)]+g_I[i+2-(aux*3)]+g_I[i+3-(aux*3)]+g_I[i+4-(aux*3)]+g_I[i+5-(aux*3)]+g_I[i+6-(aux*3)]+g_I[i+7-(aux*3)]+
            g_I[i+aux]+g_I[i-1+aux]+g_I[i-2+aux]+g_I[i-3+aux]+g_I[i-4+aux]+g_I[i+1+aux]+g_I[i+2+aux]+g_I[i+3+aux]+g_I[i+4+aux]+g_I[i+5+aux]+g_I[i+6+aux]+g_I[i+7+aux]+
            g_I[i+(aux*2)]+g_I[i-1+(aux*2)]+g_I[i-2+(aux*2)]+g_I[i-3+(aux*2)]+g_I[i-4+(aux*2)]+g_I[i+1+(aux*2)]+g_I[i+2+(aux*2)]+g_I[i+3+(aux*2)]+g_I[i+4+(aux*2)]+g_I[i+5+(aux*2)]+g_I[i+6+(aux*2)]+g_I[i+7+(aux*2)]+
            g_I[i+(aux*3)]+g_I[i-1+(aux*3)]+g_I[i-2+(aux*3)]+g_I[i-3+(aux*3)]+g_I[i-4+(aux*3)]+g_I[i+1+(aux*3)]+g_I[i+2+(aux*3)]+g_I[i+3+(aux*3)]+g_I[i+4+(aux*3)]+g_I[i+5+(aux*3)]+g_I[i+6+(aux*3)]+g_I[i+7+(aux*3)]+
 g_I[i+(aux*4)]+g_I[i-1+(aux*4)]+g_I[i-2+(aux*4)]+g_I[i-3+(aux*4)]+g_I[i-4+(aux*4)]+g_I[i-5+(aux*4)]+g_I[i-6+(aux*4)]+g_I[i-7+(aux*4)]
+g_I[i+1+(aux*4)]+g_I[i+2+(aux*4)]+g_I[i+3+(aux*4)]+g_I[i+4+(aux*4)] +g_I[i+5+(aux*4)] +g_I[i+6+(aux*4)]
+ g_I[i-(aux*4)]+g_I[i-1-(aux*4)]+g_I[i-2-(aux*4)]+g_I[i-3-(aux*4)]+g_I[i-4-(aux*4)]+g_I[i-5-(aux*4)]+g_I[i-6-(aux*4)]+g_I[i-7-(aux*4)]
+g_I[i+1-(aux*4)]+g_I[i+2-(aux*4)]+g_I[i+3-(aux*4)]+g_I[i+4-(aux*4)] +g_I[i+5-(aux*4)] +g_I[i+6-(aux*4)] +g_I[i+7-(aux*4)]  +
 g_I[i+(aux*5)]+g_I[i-1+(aux*5)]+g_I[i-2+(aux*5)]+g_I[i-3+(aux*5)]+g_I[i-4+(aux*5)]+g_I[i-5+(aux*5)]+g_I[i-6+(aux*5)]+g_I[i-7+(aux*5)]
+g_I[i+1+(aux*5)]+g_I[i+2+(aux*5)]+g_I[i+3+(aux*5)]+g_I[i+4+(aux*5)] +g_I[i+5+(aux*5)]+g_I[i+6+(aux*5)]+g_I[i+7+(aux*5)]
+ g_I[i-(aux*5)]+g_I[i-1-(aux*5)]+g_I[i-2-(aux*5)]+g_I[i-3-(aux*5)]+g_I[i-4-(aux*5)]+g_I[i-5-(aux*5)]+g_I[i+1-(aux*5)]+g_I[i+2-(aux*5)]+g_I[i+3-(aux*5)]+g_I[i+4-(aux*5)] +g_I[i+5-(aux*5)]+g_I[i+6-(aux*5)]+g_I[i+7-(aux*5)]+
g_I[i+(aux*5)]+g_I[i-1+(aux*5)]+g_I[i-2+(aux*5)]+g_I[i-3+(aux*5)]+g_I[i-4+(aux*5)]+g_I[i-5+(aux*5)]+g_I[i-6+(aux*5)]+g_I[i-7+(aux*5)]
+g_I[i+1+(aux*6)]+g_I[i+2+(aux*6)]+g_I[i+3+(aux*6)]+g_I[i+4+(aux*6)] +g_I[i+5+(aux*6)]+g_I[i+6+(aux*6)]+g_I[i+7+(aux*6)]
+ g_I[i-(aux*6)]+g_I[i-1-(aux*6)]+g_I[i-2-(aux*6)]+g_I[i-3-(aux*6)]+g_I[i-4-(aux*6)]+g_I[i-5-(aux*6)]+g_I[i+1-(aux*6)]+g_I[i+2-(aux*6)]+g_I[i+3-(aux*6)]+g_I[i+4-(aux*6)] +g_I[i+5-(aux*6)]+g_I[i+6-(aux*6)]+g_I[i+7-(aux*6)]

+g_I[i+1+(aux*7)]+g_I[i+2+(aux*7)]+g_I[i+3+(aux*7)]+g_I[i+4+(aux*7)] +g_I[i+5+(aux*7)]+g_I[i+6+(aux*7)]+g_I[i+7+(aux*7)]
+ g_I[i-(aux*7)]+g_I[i-1-(aux*7)]+g_I[i-2-(aux*7)]+g_I[i-3-(aux*7)]+g_I[i-4-(aux*7)]+g_I[i-5-(aux*7)]+g_I[i+1-(aux*7)]+g_I[i+2-(aux*7)]+g_I[i+3-(aux*7)]+g_I[i+4-(aux*7)] +g_I[i+5-(aux*7)]+g_I[i+6-(aux*7)]+g_I[i+7-(aux*7)]
    )/225;




	


   	 }
	
	}//END FOR
	  // printf("TERMINA1   red %d  green %d blue %d\n", r_I[368676], g_I[368676],b_I[368676] );
	__syncthreads();
}

int main(int argc, char *argv[]) {


    if(argc < 4){
      printf("Por favor ingresar datos así: nombreimagen.png nuevaimagen.png #kernel #hilos #bloques\n");
      exit(0);}
	read_png_file(argv[1]);

	png_byte* row;
	png_byte desrow;
	png_byte desrow2;
	png_byte desrow3;
    png_byte* wrow;
    //int totalP = *width * *height;
    int numthreads = atoi(argv[4]);
    //int numblocks = atoi(argv[5]);
    char *res = (char*) malloc(30);
    int totalPixels = width * height;
    int x;
    int inputKernel = atoi(argv[3]);
     
    int kernel = inputKernel/2;
    int divi, begin, end, begin2, end2,tnum,id, p,fin;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size

    size_t size = totalPixels * sizeof(float);

    // Allocate the host input vector R
    int *h_rI = (int *)malloc(size);

    // Allocate the host input vector G
    int *h_gI = (int *)malloc(size);

    // Allocate the host input vector B
    int *h_bI = (int *)malloc(size);

   

    // Verify that allocations succeeded
    if (h_rI == NULL || h_gI == NULL || h_bI == NULL )
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    x =0;
	for(int c=0; c<height; c++) {
		row = rowPointer[c];
		for(int d=0; d<width; d++){
			wrow = &(row[d*4]);
			desrow = wrow[0];
			desrow2 = wrow[1];
			desrow3 = wrow[2];
			h_rI[x] = desrow;
			h_gI[x] = desrow2;
			h_bI[x] = desrow3;

			//printf("%d %d %d\n", r[x], g[x],b[x] );

			// desrow = g[x];
			// desrow2 = b[x];
			// desrow3 = r[x];
			// wrow[0] = desrow;
			// wrow[1] = desrow2;
			// wrow[2] = desrow3;
			// row[d*4] = *wrow;
			x++;
		}
	}

    // Allocate the device input vector R
    int *d_rI = NULL;
    err = hipMalloc((void **)&d_rI, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector r (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector G
    int *d_gI = NULL;
    err = hipMalloc((void **)&d_gI, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector g (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    int *d_bI = NULL;
    err = hipMalloc((void **)&d_bI, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector R
    int *d_rO = NULL;
    err = hipMalloc((void **)&d_rO, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector r (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector G
    int *d_gO = NULL;
    err = hipMalloc((void **)&d_gO, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector g (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector B
    int *d_bO = NULL;
    err = hipMalloc((void **)&d_bO, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector b (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    //printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_rI, h_rI, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector r from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_gI, h_gI, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector g from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_bI, h_bI, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector b from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }




divi = (height*width/numthreads) ;
	float rest = height%numthreads;
		//printf("divi %i  rest %f height %i \n", divi,rest,height);
	end2 = divi;
	if (inputKernel == 3){begin2 = 1;
	  p = 1;
	   
	 }
	 if(inputKernel == 5){begin2 = 2;
	   p = 2;
	  
	}
	if(inputKernel == 7){begin2= 3;
		  p = 3;
		   
	}
	if(inputKernel == 9){begin2= 4;

		  p = 4;
		   
	}
	if(inputKernel == 11){begin2= 5;
		  p = 5;
		   
	}
	if(inputKernel == 13){begin2= 6;
		  p = 6;
		   
	}
	if(inputKernel == 15){
						begin2= 7;
		  p = 7;
		   
	  }
	fin = begin2;
	if (numthreads == 	1 ){
		 end2 = end2 - 7;
	}





    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = numthreads;
    int blocksPerGrid =(totalPixels + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);


        myBlur3<<<1, numthreads>>>(d_rI, d_gI, d_bI, totalPixels, height,numthreads,begin2,end2,p,inputKernel,width);
    //myBlur<<<blocksPerGrid, threadsPerBlock>>>(d_rI, d_gI, d_bI, d_rO, d_gO, d_bO, totalPixels, kernel);
    err = hipGetLastError();



















    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to launch myBlur kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



// printf("XX 2  red %d  green %d blue %d\n", h_rI[368676], h_gI[368676],h_bI[368676] );

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");

    err = hipMemcpy(h_rI, d_rI, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector rI from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_gI, d_gI, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector gI from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(h_bI, d_bI, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector bI from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


 //printf("TERNA  red %d  green %d blue %d\n", h_rI[368676], h_gI[368676],h_bI[368676] );




    x =0;
	for(int c=0; c<height; c++) {
		row = rowPointer[c];
		for(int d=0; d<width; d++){
			wrow = &(row[d*4]);
			// desrow = wrow[0];
			// desrow2 = wrow[1];
			// desrow3 = wrow[2];
			// h_rI[x] = desrow;
			// h_gI[x] = desrow2;
			// h_bI[x] = desrow3;

			//printf("%d %d %d\n", r[x], g[x],b[x] );

			desrow = h_rI[x];
			desrow2 = h_gI[x];
			desrow3 = h_bI[x];
			wrow[0] = desrow;
			wrow[1] = desrow2;
			wrow[2] = desrow3;
			row[d*4] = *wrow;
			x++;
		}
	}

    //printf("Test PASSED\n");




	write_png_file(argv[2]);





    // Free device global memory
    err = hipFree(d_rI);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector rI (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_gI);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector gI (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_bI);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device vector bI (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Free host memory
    free(h_rI);
    free(h_gI);
    free(h_bI);


	return(0);
}
